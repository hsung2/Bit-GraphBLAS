/*
* cuSPARSE using CUDA 11.0 on A100 baseline
*/

#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 5
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include <vector>
#include "backend/readMtx.hpp"
#include "backend/csr2bsr_batch_bsrbmv.cu"

/// ======================
// csr metadata
int nrows, ncols, nnz;

// csr host
int *h_csrRowPtr, *h_csrColInd;
float *h_csrVal;

// csr device
int *csrRowPtr, *csrColInd;
float *csrVal;

// vec host
float *B;

// vec device
float *fB;

// cuSPARSE vec
float *dX, *dY;

// cusparse handles
hipsparseHandle_t handle_csr2csc;

hipsparseHandle_t handle_csr;
hipsparseStatus_t cusparse_status;

hipsparseSpMatDescr_t mat_A;
hipsparseDnVecDescr_t vecX;
hipsparseDnVecDescr_t vecY;

/// ======================
void readMtxCSR(const char *filename, bool transpose = false)
{
    // graphblast mmio interface
    std::vector<int> row_indices;
    std::vector<int> col_indices;
    std::vector<float> values;
    char *dat_name;
    readMtx(filename, &row_indices, &col_indices, &values,
            &nrows, &ncols, &nnz, 0, false, &dat_name); // directed, mtxinfo

    h_csrRowPtr = (int *)malloc(sizeof(int) * (nrows + 1));
    h_csrColInd = (int *)malloc(sizeof(int) * nnz);
    h_csrVal = (float *)malloc(sizeof(float) * nnz);
    coo2csr(h_csrRowPtr, h_csrColInd, h_csrVal,
            row_indices, col_indices, values, nrows, ncols);

    // copy csr to device
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows + 1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows + 1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);

    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1, 1>>>(csrVal, nnz, 1.0);
}

void genRandVec()
{
    // generate random vector
    srand(time(0));
    B = (float *)malloc(nrows * sizeof(float));
    for (int i = 0; i < nrows; i++)
    {
        float x = (float)rand() / RAND_MAX;
        B[i] = (x > 0.5) ? 1 : 0;
    }
}

double evalCSRSpmvFloatCuSPARSE()
{
    // metadata for cuSPARSE API
    hipsparseCreate(&handle_csr);

    // dummy multiplication variables
    // y = α ∗ op ( A ) ∗ x + β ∗ y
#if TEST_TIMES > 1
    float alpha = 1.0, beta = 1.0;
#else
    float alpha = 1.0, beta = 0.0;
#endif

    // create CSR
    hipsparseCreateCsr(&mat_A, nrows, ncols, nnz,
                      csrRowPtr, csrColInd, csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // create dense vector storage
    hipMalloc((void **)&dX, sizeof(float) * nrows);
    hipMemcpy(dX, B, sizeof(float) * nrows, hipMemcpyHostToDevice); // do not move paddings
    hipMalloc((void **)&dY, sizeof(float) * nrows);
    setDeviceValArr<int, float><<<1, 1>>>(dY, nrows, 0);

    hipsparseCreateDnVec(&vecX, nrows, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, nrows, dY, HIP_R_32F);

    // buffer
    void *buffer;
    size_t tempInt;
    hipsparseSpMV_bufferSize(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, mat_A, vecX, &beta, vecY,
                            HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &tempInt);
    hipMalloc(&buffer, tempInt);

    // ------
    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i = 0; i < TEST_TIMES; i++)
    {
        hipsparseSpMV(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, mat_A, vecX, &beta, vecY,
                     HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &buffer);
    }

    csr_timer.Stop();
    double cusparsecsrspmvfloat_time = csr_timer.ElapsedMillis() / double(TEST_TIMES);
    // ------

    // free temp storage
    hipFree(buffer);

    return cusparsecsrspmvfloat_time;
}

void freeCSR()
{
    // free cusparse csr spmv
    hipsparseDestroySpMat(mat_A);
    hipsparseDestroy(handle_csr);

    // free csr mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    // free vec mem
    hipFree(dX);
    hipFree(dY);
}

/// ======================
void verifyResult()
{
    // copy result to host for verification
    float *Y;
    hipMalloc((void **)&Y, sizeof(float) * nrows);
    float *result_cusparsecsrspmvfloat = (float *)malloc(sizeof(float) * nrows);
    hipsparseDnVecGetValues(vecY, (void **)&Y);
    hipMemcpy(result_cusparsecsrspmvfloat, Y, sizeof(float) * nrows, hipMemcpyDeviceToHost);
    hipFree(Y);

    // verify bsrbmv with cuSPARSE baseline
    for (int i = 0; i < nrows; i++)
        printf("%f ", result_cusparsecsrspmvfloat[i]);
    printf("\n");

    // free mem
    free(result_cusparsecsrspmvfloat);
}

/// ======================
int main(int argc, char *argv[])
{
    char *filename = argv[1];      // e.g. "G43.mtx"
    int transpose = atoi(argv[2]); // 1: transpose, 0: default

    // bmv: C = A * B
    // init
    hipSetDevice(0);
    readMtxCSR(filename, transpose);
    genRandVec();

    // csr spmv
    double time = evalCSRSpmvFloatCuSPARSE();

    // verify
    // verifyResult();

    // free mem
    freeCSR();

    // print result
    printf("%f ", time);
}
